#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "PJ3.h"

#define blockSize 32

template <int BLOCK_SIZE> __global__ void MatMulKernel2DAnySize(double* C, double* A, double* B, int wA, int wC, int hC)
{
    int wB = wC;
    int maxIdxA = wA * hC;

    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;
    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    while (wA * BLOCK_SIZE * by < maxIdxA) {
        // Index of the first sub-matrix of A processed by the block
        int aBegin = wA * BLOCK_SIZE * by;

        // Index of the last sub-matrix of A processed by the block
        int aEnd = aBegin + wA - 1;

        // Step size used to iterate through the sub-matrices of A
        int aStep = BLOCK_SIZE;

        // Index of the first sub-matrix of B processed by the block
        int bBegin = BLOCK_SIZE * bx;

        // Step size used to iterate through the sub-matrices of B
        int bStep = BLOCK_SIZE * wB;

        // Csub is used to store the element of the block sub-matrix
        // that is computed by the thread
        double Csub = 0;

        // Loop over all the sub-matrices of A and B
        // required to compute the block sub-matrix
        int flag = 0;
        for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) {
            // Declaration of the shared memory array As used to
            // store the sub-matrix of A
            __shared__ double As[BLOCK_SIZE][BLOCK_SIZE];

            // Declaration of the shared memory array Bs used to
            // store the sub-matrix of B
            __shared__ double Bs[BLOCK_SIZE][BLOCK_SIZE];

            // Load the matrices from device memory
            // to shared memory; each thread loads
            // one element of each matrix
            if (flag * BLOCK_SIZE + tx < wA || flag * BLOCK_SIZE + ty < hC) {
                As[ty][tx] = A[a + wA * ty + tx];
            }
            else {
                As[ty][tx] = 0.0;
            }

            if (flag * BLOCK_SIZE + ty < wA || flag * BLOCK_SIZE + tx < wC) {
                Bs[ty][tx] = B[b + wB * ty + tx];
            }
            else {
                Bs[ty][tx] = 0.0;
            }

            // Synchronize to make sure the matrices are loaded
            __syncthreads();

            // Multiply the two matrices together;
            // each thread computes one element
            // of the block sub-matrix
#pragma unroll

            for (int k = 0; k < BLOCK_SIZE; ++k) {
                Csub += As[ty][k] * Bs[k][tx];
            }

            // Synchronize to make sure that the preceding
            // computation is done before loading two new
            // sub-matrices of A and B in the next iteration
            __syncthreads();
            flag++;
        }

        // Write the block sub-matrix to device memory;
        // each thread writes one element
        if (BLOCK_SIZE * bx + tx < wC && BLOCK_SIZE * by + ty < hC) { // thread could over max.
            C[wB * BLOCK_SIZE * by + BLOCK_SIZE * bx + wB * ty + tx] = Csub;
        }
        bx += BLOCK_SIZE;
        by += BLOCK_SIZE;
    }
}

void Matrix_Mul(int N, double h_A[],double h_B[],double h_C[]){
    
    // Allocate device memory
    double* d_A, * d_B, * d_C;

    // Allocate host matrix C
    unsigned int mem_size_C = N * N * sizeof(double);

    if (h_C == NULL) {
        fprintf(stderr, "Failed to allocate host matrix C!\n");
        exit(EXIT_FAILURE);
    }

    hipMalloc(reinterpret_cast<void**>(&d_A), mem_size_C);
    hipMalloc(reinterpret_cast<void**>(&d_B), mem_size_C);
    hipMalloc(reinterpret_cast<void**>(&d_C), mem_size_C);
 

    // copy host memory to device
    hipMemcpy(d_A, h_A, mem_size_C, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, mem_size_C, hipMemcpyHostToDevice);

    // Setup execution parameters
    dim3 threads(blockSize, blockSize);
    dim3 grid;

    // select diff blocks for kerenl
    void (*MMKernel2DAnySizeExe)(double*, double*, double*, int, int, int);
    if (blockSize <= 16) {
        MMKernel2DAnySizeExe = MatMulKernel2DAnySize<16>;
    }
    else {
        MMKernel2DAnySizeExe = MatMulKernel2DAnySize<32>;
    }

    grid = dim3(N / threads.x+1, N / threads.y+1);
    MMKernel2DAnySizeExe << <grid, threads>> > (d_C, d_A, d_B, N, N, N);

    // Copy result from device to host
    hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost);

    // Clean up memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}